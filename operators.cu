
#include <hip/hip_runtime.h>

inline __host__ __device__ float3 operator-(const float3 &a, const float3 &b) {
  return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}
inline __host__ __device__ float3 operator+(const float3 &a, const float3 &b) {
  return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}
inline __host__ __device__ float3 operator/(float3 a, float3 b)
{
    return make_float3(a.x / b.x, a.y / b.y, a.z / b.z);
}
inline __host__ __device__ float3 operator/(float3 a, float b)
{
    return make_float3(a.x / b, a.y / b, a.z / b);
}
inline __host__ __device__ float3 operator/(float b, float3 a)
{
    return make_float3(b / a.x, b / a.y, b / a.z);
}
inline __host__ __device__ float3 operator*(float3 a, float3 b)
{
    return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}
inline __host__ __device__ float3 operator*(float3 a, float b)
{
    return make_float3(a.x * b, a.y * b, a.z * b);
}
inline __host__ __device__ float3 operator*(float b, float3 a)
{
    return make_float3(b * a.x, b * a.y, b * a.z);
}
inline __host__ __device__ float3 operator-(float3 &a)
{
    return make_float3(-a.x, -a.y, -a.z);
}
inline __host__ __device__ int3 operator-(int3 a, int b)
{
    return make_int3(a.x - b, a.y - b, a.z - b);
}
inline __host__ __device__ float3 fmaxf(float3 a, float3 b)
{
	return make_float3(fmaxf(a.x,b.x), fmaxf(a.y,b.y), fmaxf(a.z,b.z));
}
inline __host__ __device__ float3 fminf(float3 a, float3 b)
{
	return make_float3(fminf(a.x,b.x), fminf(a.y,b.y), fminf(a.z,b.z));
}
inline __host__ __device__ int3 max(int3 a, int3 b)
{
    return make_int3(max(a.x,b.x), max(a.y,b.y), max(a.z,b.z));
}
inline __host__ __device__ int3 operator+(int3 a, int b)
{
    return make_int3(a.x + b, a.y + b, a.z + b);
}
inline __host__ __device__ uint3 operator+(uint3 a, uint3 b)
{
    return make_uint3(a.x + b.x, a.y + b.y, a.z + b.z);
}
inline __host__ __device__ int3 min(int3 a, int3 b)
{
    return make_int3(min(a.x,b.x), min(a.y,b.y), min(a.z,b.z));
}